#include "hip/hip_runtime.h"
#include "benchmark.h"
#include "nbody_cuda.cuh"
#include "structures.h"
#include "timer.h"
#include <cmath>
#include <cstddef>
#include <iterator>

#define BLOCK_SIZE 256
#define nStreams 12

// constant memory
// mimic main.cpp defaults
__constant__ float G = 1;
__constant__ float dt = 0.001;
__constant__ float theta_sq = 0.8f * 0.8f;
__constant__ float eps_sq = 0.05f * 0.05f;

int numBlocks;
int totalPairs;
int numPairBlocks;

int updateChunks;
int updateBlocks;

body_t *d_bodies;

node_t *d_nodes;
octree_t *d_octree;
bool bh_setup = false;

hipStream_t streams[nStreams];

__global__ void naive_kernel(int pointCount, body_t *bodies) {
  int particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (particle_idx >= pointCount)
    return;

  float4 particle = bodies[particle_idx].position; // Load into register

  __shared__ float4 shared_pos[BLOCK_SIZE];
  for (int i = 0; i < pointCount; i += BLOCK_SIZE) {
    int other_idx = i + threadIdx.x;

    if (other_idx >= pointCount)
      return;
    shared_pos[threadIdx.x] = bodies[other_idx].position;

    __syncthreads();

#pragma unroll
    for (int j = 0; j < BLOCK_SIZE; j++) {
      if (i + j >= pointCount)
        break;

      float4 other = shared_pos[j]; // Load into register
      float dx = other.x - particle.x;
      float dy = other.y - particle.y;
      float dz = other.z - particle.z;

      float distSq =
          dx * dx + dy * dy + dz * dz + eps_sq; // Avoid small distances
      float invDist = rsqrtf(distSq);
      float invDist3 = invDist * invDist * invDist;
      float force = G * particle.w * other.w * invDist3;
      float fw = force / particle.w;
      bodies[particle_idx].velocity.x += dx * fw * dt;
      bodies[particle_idx].velocity.y += dy * fw * dt;
      bodies[particle_idx].velocity.z += dz * fw * dt;
    }
    __syncthreads();
  }
}

__global__ void bh_kernel(body_t *bodies, octree_t *octree) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int node = ROOT;
  float3 acceleration = {0, 0, 0};
  float4 position = bodies[tid].position;

  while (true) {
    node_t n = octree->nodes[node];
    float dx = n.center_of_mass.x - position.x;
    float dy = n.center_of_mass.y - position.y;
    float dz = n.center_of_mass.z - position.z;
    float d_sq = dx * dx + dy * dy + dz * dz + eps_sq;

    if (4 * n.box.half_extent * n.box.half_extent < theta_sq * d_sq) {
      float inv_d = rsqrtf(d_sq);
      float acc = G * n.center_of_mass.w * inv_d * inv_d * inv_d;

      acceleration.x += dx * acc;
      acceleration.y += dy * acc;
      acceleration.z += dz * acc;

      if (n.next == ROOT) {
        break;
      }
      node = n.next;
      __syncthreads();
    } else if (n.children == ROOT) {
      for (int j = n.pos_idx; j < n.pos_idx + n.count; j++) {
        float4 other = bodies[j].position;
        float dx = other.x - position.x;
        float dy = other.y - position.y;
        float dz = other.z - position.z;
        float d_sq = dx * dx + dy * dy + dz * dz + eps_sq;

        float inv_d = rsqrtf(d_sq);
        float acc = G * other.w * inv_d * inv_d * inv_d;

        acceleration.x += dx * acc;
        acceleration.y += dy * acc;
        acceleration.z += dz * acc;
      }
      if (n.next == ROOT) {
        break;
      }
      node = n.next;
      __syncthreads();
    } else {
      node = n.children;
      __syncthreads();
    }
  }

  atomicAdd(&bodies[tid].velocity.x, acceleration.x * dt);
  atomicAdd(&bodies[tid].velocity.y, acceleration.y * dt);
  atomicAdd(&bodies[tid].velocity.z, acceleration.z * dt);
}

__global__ void update_pos_kernel(int pointCount, body_t *bodies) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < pointCount) {
    atomicAdd(&bodies[i].position.x, bodies[i].velocity.x);
    atomicAdd(&bodies[i].position.y, bodies[i].velocity.y);
    atomicAdd(&bodies[i].position.z, bodies[i].velocity.z);
  }
  __syncthreads();
}

void gpu_update_position(int N, body_t *bodies) {
  BENCHMARK_START("BodiesD2H");
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * updateChunks;
    int currentChunkSize = std::min(updateChunks, N - offset);

    if (currentChunkSize > 0) {
      update_pos_kernel<<<numBlocks, BLOCK_SIZE, 0, streams[i]>>>(
          currentChunkSize, d_bodies + offset);
      cudaCheckErrors("UPDATE Kernel execution failed");

      hipMemcpyAsync(&(bodies[offset]), d_bodies + offset,
                      currentChunkSize * sizeof(body_t), hipMemcpyDeviceToHost,
                      streams[i]);
    }
  }

  for (int i = 0; i < nStreams; ++i) {
    hipStreamSynchronize(streams[i]);
  }
  BENCHMARK_STOP("BodiesD2H");
}

void gpu_pin_mem(int N, body_t *bodies) {
  hipHostMalloc(&bodies, N * sizeof(body_t)); // pin host mem
}

void gpu_setup(int N, body_t *bodies) {
  // kernel dims
  numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
  totalPairs = (N * (N - 1)) / 2;
  numPairBlocks = (totalPairs + BLOCK_SIZE - 1) / BLOCK_SIZE;

  updateChunks = (N + nStreams - 1) / nStreams;
  updateBlocks = (updateChunks + BLOCK_SIZE - 1) / BLOCK_SIZE;

  // allocate and cpy bodies to device
  hipMalloc(&d_bodies, N * sizeof(body_t));
  hipMemcpy(d_bodies, bodies, N * sizeof(body_t), hipMemcpyHostToDevice);

  // create copy device to host streams
  for (int i = 0; i < nStreams; ++i) {
    hipStreamCreate(&streams[i]);
  }
}

void gpu_update_naive(int N, body_t *bodies) {
  BENCHMARK_START("UpdateNaive_GPU");
  naive_kernel<<<numPairBlocks, BLOCK_SIZE>>>(N, d_bodies);
  hipDeviceSynchronize();
  BENCHMARK_STOP("UpdateNaive_GPU");
  cudaCheckErrors("STEP Kernel execution failed");
  gpu_update_position(N, bodies);
}

void gpu_cleanup_naive(body_t *bodies) {
  hipFree(d_bodies);
  hipHostFree(bodies);

  // destroy streams
  for (int i = 0; i < nStreams; ++i) {
    hipStreamDestroy(streams[i]);
  }
}

void gpu_setup_bh(body_t *bodies, octree_t *octree, int N) {
  hipMalloc(&d_octree, sizeof(octree_t));
  hipMalloc(&d_nodes, octree->max_nodes * sizeof(node_t));

  octree_t h_octree = *octree;
  h_octree.nodes = d_nodes; // Update to device pointer
  hipMemcpy(d_octree, &h_octree, sizeof(octree_t), hipMemcpyHostToDevice);
}

void gpu_update_bh(int N, body_t *bodies, octree_t *octree) {
  if (!bh_setup) {
    gpu_setup_bh(bodies, octree, N);
    bh_setup = true;
  }

  BENCHMARK_START("OctreeH2D");
  hipMemcpy(d_nodes, octree->nodes, octree->max_nodes * sizeof(node_t),
             hipMemcpyHostToDevice);
  BENCHMARK_STOP("OctreeH2D");

  BENCHMARK_START("UpdateBH_GPU");
  bh_kernel<<<numBlocks, BLOCK_SIZE>>>(d_bodies, d_octree);
  hipDeviceSynchronize();
  BENCHMARK_STOP("UpdateBH_GPU");
  cudaCheckErrors("STEP Kernel execution failed");

  gpu_update_position(N, bodies);
}

void gpu_cleanup_bh(body_t *bodies) {
  hipFree(d_bodies);
  hipHostFree(bodies);
  hipFree(d_octree);
  hipFree(d_nodes);

  // Destroy streams
  for (int i = 0; i < nStreams; ++i) {
    hipStreamDestroy(streams[i]);
  }
}
